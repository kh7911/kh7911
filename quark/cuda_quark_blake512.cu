#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#define ROTR(x,n) ROTR64(x,n)

#define USE_SHUFFLE 0

// die Message it Padding zur Berechnung auf der GPU
__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

// ---------------------------- BEGIN CUDA quark_blake512 functions ------------------------------------

__constant__ uint8_t c_sigma[16][16];

const uint8_t host_sigma[16][16] =
{
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
  {12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
  {13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
  { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
  {10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__
const uint64_t c_u512[16] =
{
  0x243f6a8885a308d3ULL, 0x13198a2e03707344ULL, 
  0xa4093822299f31d0ULL, 0x082efa98ec4e6c89ULL,
  0x452821e638d01377ULL, 0xbe5466cf34e90c6cULL, 
  0xc0ac29b7c97c50ddULL, 0x3f84d5b5b5470917ULL,
  0x9216d5d98979fb1bULL, 0xd1310ba698dfb5acULL, 
  0x2ffd72dbd01adfb7ULL, 0xb8e1afed6a267e96ULL,
  0xba7c9045f12c7f99ULL, 0x24a19947b3916cf7ULL, 
  0x0801f2e2858efc16ULL, 0x636920d871574e69ULL
};

#define G(a,b,c,d,x) { \
	uint32_t idx1 = sigma[i][x]; \
	uint32_t idx2 = sigma[i][x+1]; \
	v[a] += (m[idx1] ^ u512[idx2]) + v[b]; \
	v[d] = ROTR( v[d] ^ v[a], 32); \
	v[c] += v[d]; \
	v[b] = ROTR( v[b] ^ v[c], 25); \
	v[a] += (m[idx2] ^ u512[idx1]) + v[b]; \
	v[d] = ROTR( v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR( v[b] ^ v[c], 11); \
}

__device__ static
void quark_blake512_compress( uint64_t *h, const uint64_t *block, const uint8_t ((*sigma)[16]), const uint64_t *u512, const int T0)
{
    uint64_t v[16], m[16], i;

	#pragma unroll 16
	for( i = 0; i < 16; i++) {
		m[i] = cuda_swab64(block[i]);
	}

	#pragma unroll 8
	for (i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = u512[0];
	v[ 9] = u512[1];
	v[10] = u512[2];
	v[11] = u512[3];
	v[12] = u512[4] ^ T0;
	v[13] = u512[5] ^ T0;
	v[14] = u512[6];
	v[15] = u512[7];

	//#pragma unroll 16
	for( i = 0; i < 16; ++i )
	{
		/* column step */
		G( 0, 4, 8, 12, 0 );
		G( 1, 5, 9, 13, 2 );
		G( 2, 6, 10, 14, 4 );
		G( 3, 7, 11, 15, 6 );
		/* diagonal step */
		G( 0, 5, 10, 15, 8 );
		G( 1, 6, 11, 12, 10 );
		G( 2, 7, 8, 13, 12 );
		G( 3, 4, 9, 14, 14 );
	}

	#pragma unroll 16
	for( i = 0; i < 16; ++i )
		h[i % 8] ^= v[i];
}

// Hash-Padding
__device__ __constant__
static const uint64_t d_constHashPadding[8] = {
	0x0000000000000080ull,
	0,
	0,
	0,
	0,
	0x0100000000000000ull,
	0,
	0x0002000000000000ull
};

__global__ __launch_bounds__(256, 4)
void quark_blake512_gpu_hash_64(int threads, uint32_t startNounce, uint32_t *g_nonceVector, uint64_t *g_hash)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);

#if USE_SHUFFLE
	const int warpID = threadIdx.x & 0x0F; // 16 warps
	const int warpBlockID = (thread + 15)>>4; // aufrunden auf volle Warp-Bl�cke
	const int maxHashPosition = thread<<3;
#endif

#if USE_SHUFFLE
	if (warpBlockID < ( (threads+15)>>4 ))
#else
	if (thread < threads)
#endif
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[hashPosition<<3]; // hashPosition * 8

		// 128 Bytes
		uint64_t buf[16];

		// State
		uint64_t h[8] = {
			0x6a09e667f3bcc908ULL,
			0xbb67ae8584caa73bULL,
			0x3c6ef372fe94f82bULL,
			0xa54ff53a5f1d36f1ULL,
			0x510e527fade682d1ULL,
			0x9b05688c2b3e6c1fULL,
			0x1f83d9abfb41bd6bULL,
			0x5be0cd19137e2179ULL
		};

		// Message for first round
		#pragma unroll 8
		for (int i=0; i < 8; ++i)
			buf[i] = inpHash[i];

		#pragma unroll 8
		for (int i=0; i < 8; i++)
			buf[i+8] = d_constHashPadding[i];

		// Ending round
		quark_blake512_compress( h, buf, c_sigma, c_u512, 512 );

#if __CUDA_ARCH__ <= 350
		uint32_t *outHash = (uint32_t*)&g_hash[8 * hashPosition];
		#pragma unroll 8
		for (int i=0; i < 8; i++) {
			outHash[2*i+0] = cuda_swab32( _HIWORD(h[i]) );
			outHash[2*i+1] = cuda_swab32( _LOWORD(h[i]) );
		}
#else
		uint64_t *outHash = &g_hash[8 * hashPosition];
		for (int i=0; i < 8; i++) {
			outHash[i] = cuda_swab64(h[i]);
		}
#endif
	}
}

__global__ void quark_blake512_gpu_hash_80(int threads, uint32_t startNounce, void *outputHash)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t buf[16];
		uint32_t nounce = startNounce + thread;

		uint64_t h[8] = {
			0x6a09e667f3bcc908ULL,
			0xbb67ae8584caa73bULL,
			0x3c6ef372fe94f82bULL,
			0xa54ff53a5f1d36f1ULL,
			0x510e527fade682d1ULL,
			0x9b05688c2b3e6c1fULL,
			0x1f83d9abfb41bd6bULL,
			0x5be0cd19137e2179ULL
		};

		// Message f�r die erste Runde in Register holen
		#pragma unroll 16
		for (int i=0; i < 16; ++i)
			buf[i] = c_PaddedMessage80[i];

		// The test Nonce
		((uint32_t*)buf)[19] = cuda_swab32(nounce);

		quark_blake512_compress( h, buf, c_sigma, c_u512, 640 );

#if __CUDA_ARCH__ <= 350
		uint32_t *outHash = (uint32_t *)outputHash + 16 * thread;
		#pragma unroll 8
		for (uint32_t i=0; i < 8; i++) {
			outHash[2*i]   = cuda_swab32( _HIWORD(h[i]) );
			outHash[2*i+1] = cuda_swab32( _LOWORD(h[i]) );
		}
#else
		uint64_t *outHash = (uint64_t *)outputHash + 8 * thread;
		for (uint32_t i=0; i < 8; i++) {
			outHash[i] = cuda_swab64( h[i] );
		}
#endif

	}
}


// ---------------------------- END CUDA quark_blake512 functions ------------------------------------

// Setup-Funktionen
__host__ void quark_blake512_cpu_init(int thr_id, int threads)
{
	// Kopiere die Hash-Tabellen in den GPU-Speicher
	hipMemcpyToSymbol(HIP_SYMBOL( c_sigma),
						host_sigma,
						sizeof(host_sigma),
						0, hipMemcpyHostToDevice);
}

// Blake512 f�r 80 Byte grosse Eingangsdaten
__host__ void quark_blake512_cpu_setBlock_80(void *pdata)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	PaddedMessage[80] = 0x80;
	PaddedMessage[111] = 1;
	PaddedMessage[126] = 0x02;
	PaddedMessage[127] = 0x80;

	CUDA_SAFE_CALL(
		hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice)
	);
}

__host__ void quark_blake512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash, int order)
{
	const int threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	quark_blake512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, d_nonceVector, (uint64_t*)d_outputHash);

	// Strategisches Sleep Kommando zur Senkung der CPU Last
	MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void quark_blake512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_outputHash, int order)
{
	const int threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	quark_blake512_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, d_outputHash);

	// Strategisches Sleep Kommando zur Senkung der CPU Last
	MyStreamSynchronize(NULL, order, thr_id);
}
