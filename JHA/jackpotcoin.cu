#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_keccak.h"
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_jh.h"
#include "sph/sph_skein.h"
}

#include "miner.h"
#include "cuda_helper.h"

extern int device_map[8];

static uint32_t *d_hash[8];

extern void jackpot_keccak512_cpu_init(int thr_id, int threads);
extern void jackpot_keccak512_cpu_setBlock(void *pdata, size_t inlen);
extern void jackpot_keccak512_cpu_hash(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void quark_blake512_cpu_init(int thr_id, int threads);
extern void quark_blake512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_groestl512_cpu_init(int thr_id, int threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_jh512_cpu_init(int thr_id, int threads);
extern void quark_jh512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, int threads);
extern void quark_skein512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void jackpot_compactTest_cpu_init(int thr_id, int threads);
extern void jackpot_compactTest_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable, 
											uint32_t *d_nonces1, size_t *nrm1,
											uint32_t *d_nonces2, size_t *nrm2,
											int order);

// Speicher zur Generierung der Noncevektoren für die bedingten Hashes
static uint32_t *d_jackpotNonces[8];
static uint32_t *d_branch1Nonces[8];
static uint32_t *d_branch2Nonces[8];
static uint32_t *d_branch3Nonces[8];

// Original jackpothash Funktion aus einem miner Quelltext
extern "C" unsigned int jackpothash(void *state, const void *input)
{
    sph_blake512_context     ctx_blake;
    sph_groestl512_context   ctx_groestl;
    sph_jh512_context        ctx_jh;
    sph_keccak512_context    ctx_keccak;
    sph_skein512_context     ctx_skein;

    uint32_t hash[16];

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, input, 80);
    sph_keccak512_close(&ctx_keccak, hash);

    unsigned int round;
    for (round = 0; round < 3; round++) {
        if (hash[0] & 0x01) {
           sph_groestl512_init(&ctx_groestl);
           sph_groestl512 (&ctx_groestl, (&hash), 64);
           sph_groestl512_close(&ctx_groestl, (&hash));
        }
        else {
           sph_skein512_init(&ctx_skein);
           sph_skein512 (&ctx_skein, (&hash), 64);
           sph_skein512_close(&ctx_skein, (&hash));
        }
        if (hash[0] & 0x01) {
           sph_blake512_init(&ctx_blake);
           sph_blake512 (&ctx_blake, (&hash), 64);
           sph_blake512_close(&ctx_blake, (&hash));
        }
        else {
           sph_jh512_init(&ctx_jh);
           sph_jh512 (&ctx_jh, (&hash), 64);
           sph_jh512_close(&ctx_jh, (&hash));
        }
    }
    memcpy(state, hash, 32);

    return round;
}


extern bool opt_benchmark;

extern "C" int scanhash_jackpot(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	const uint32_t Htarg = ptarget[7];

	const int throughput = 256*4096*4; // 100;

	static bool init[8] = {0,0,0,0,0,0,0,0};
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		// Konstanten kopieren, Speicher belegen
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);
		jackpot_keccak512_cpu_init(thr_id, throughput);
		jackpot_compactTest_cpu_init(thr_id, throughput);
		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		cuda_check_cpu_init(thr_id, throughput);
		hipMalloc(&d_jackpotNonces[thr_id], sizeof(uint32_t)*throughput*2);
		hipMalloc(&d_branch1Nonces[thr_id], sizeof(uint32_t)*throughput*2);
		hipMalloc(&d_branch2Nonces[thr_id], sizeof(uint32_t)*throughput*2);
		hipMalloc(&d_branch3Nonces[thr_id], sizeof(uint32_t)*throughput*2);
		init[thr_id] = true;
	}

	uint32_t endiandata[22];
	for (int k=0; k < 22; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	jackpot_keccak512_cpu_setBlock((void*)endiandata, 80);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// erstes Keccak512 Hash mit CUDA
		jackpot_keccak512_cpu_hash(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		size_t nrm1, nrm2, nrm3;

		// Runde 1 (ohne Gröstl)

		jackpot_compactTest_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], NULL,
				d_branch1Nonces[thr_id], &nrm1,
				d_branch3Nonces[thr_id], &nrm3,
				order++);

		// verfolge den skein-pfad weiter
		quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

		// noch schnell Blake & JH
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// Runde 3 (komplett)

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_groestl512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// Runde 3 (komplett)

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_groestl512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// Scan nach Gewinner Hashes auf der GPU
		uint32_t foundNonce = cuda_check_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);
		if  (foundNonce != 0xffffffff)
		{
			unsigned int rounds;
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);

			// diese jackpothash Funktion gibt die Zahl der Runden zurück
			rounds = jackpothash(vhash64, endiandata);

			if ((vhash64[7]<=Htarg) && fulltest(vhash64, ptarget)) {

				pdata[19] = foundNonce;
				*hashes_done = (foundNonce - first_nonce + 1)/2;
				//applog(LOG_INFO, "GPU #%d: result for nonce $%08X does validate on CPU (%d rounds)!", thr_id, foundNonce, rounds);
				return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU (%d rounds)!", thr_id, foundNonce, rounds);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = (pdata[19] - first_nonce + 1)/2;
	return 0;
}
