#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Aug. 2014
 */

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
#include <stdint.h>
#include <memory.h>
}

/* threads per block */
#define TPB 128

/* crc32.c */
extern "C" uint32_t crc32_u32t(const uint32_t *buf, size_t size);

extern "C" int blake256_rounds = 14;

/* hash by cpu with blake 256 */
extern "C" void blake256hash(void *output, const void *input, int8_t rounds = 14)
{
	unsigned char hash[64];
	sph_blake256_context ctx;

	/* in sph_blake.c */
	blake256_rounds = rounds;

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);
	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

#define MAXU 0xffffffffU

// in cpu-miner.c
extern bool opt_n_threads;
extern int device_map[8];

__constant__
static uint32_t __align__(32) c_data[20];

/* 8 adapters max (-t threads) */
static uint32_t *d_resNonce[8];
static uint32_t *h_resNonce[8];

/* max count of found nounces in one call */
#define NBN 2
static uint32_t extra_results[NBN-1] = { MAXU };

#define USE_CACHE 1
/* midstate hash cache, this algo is run on 2 parts */
#if USE_CACHE
__device__ static uint32_t cache[8];
__device__ static uint32_t prevsum = 0;
#endif

/* prefer uint32_t to prevent size conversions = speed +5/10 % */
__constant__
static uint32_t __align__(32) c_sigma[16][16];
const uint32_t host_sigma[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__
static const uint32_t __align__(32) c_IV256[8] = {
	SPH_C32(0x6A09E667), SPH_C32(0xBB67AE85),
	SPH_C32(0x3C6EF372), SPH_C32(0xA54FF53A),
	SPH_C32(0x510E527F), SPH_C32(0x9B05688C),
	SPH_C32(0x1F83D9AB), SPH_C32(0x5BE0CD19)
};

__device__ __constant__
static const uint32_t __align__(32) c_u256[16] = {
	SPH_C32(0x243F6A88), SPH_C32(0x85A308D3),
	SPH_C32(0x13198A2E), SPH_C32(0x03707344),
	SPH_C32(0xA4093822), SPH_C32(0x299F31D0),
	SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
	SPH_C32(0x452821E6), SPH_C32(0x38D01377),
	SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C),
	SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD),
	SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
};

#define GS(a,b,c,d,x) { \
	const uint32_t idx1 = c_sigma[i][x]; \
	const uint32_t idx2 = c_sigma[i][x+1]; \
	v[a] += (m[idx1] ^ c_u256[idx2]) + v[b]; \
	v[d] = SPH_ROTL32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ c_u256[idx1]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
}

/* Second part (64-80) msg never change, store it */
__device__ __constant__
static const uint32_t __align__(32) c_Padding[16] = {
	0, 0, 0, 0,
	0x80000000UL, 0, 0, 0,
	0, 0, 0, 0,
	0, 1, 0, 640,
};

__device__ static
void blake256_compress(uint32_t *h, const uint32_t *block, const uint32_t T0, const int rounds)
{
	uint32_t /* __align__(8) */ m[16];
	uint32_t /* __align__(8) */ v[16];

	m[0] = block[0];
	m[1] = block[1];
	m[2] = block[2];
	m[3] = block[3];

	for (uint32_t i = 4; i < 16; i++) {
		m[i] = (T0 == 0x200) ? block[i] : c_Padding[i];
	}

	//#pragma unroll 8
	for(uint32_t i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = c_u256[0];
	v[ 9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[12] = c_u256[4] ^ T0;
	v[13] = c_u256[5] ^ T0;
	v[14] = c_u256[6];
	v[15] = c_u256[7];

	for (int i = 0; i < rounds; i++) {
		/* column step */
		GS(0, 4, 0x8, 0xC, 0x0);
		GS(1, 5, 0x9, 0xD, 0x2);
		GS(2, 6, 0xA, 0xE, 0x4);
		GS(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		GS(0, 5, 0xA, 0xF, 0x8);
		GS(1, 6, 0xB, 0xC, 0xA);
		GS(2, 7, 0x8, 0xD, 0xC);
		GS(3, 4, 0x9, 0xE, 0xE);
	}

	//#pragma unroll 16
	for (uint32_t i = 0; i < 16; i++) {
		uint32_t j = i % 8U;
		h[j] ^= v[i];
	}
}

__global__
void blake256_gpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint32_t *resNounce,
	const uint64_t highTarget, const int crcsum, const int rounds)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nounce = startNonce + thread;
		uint32_t h[8];

		#pragma unroll
		for(int i=0; i<8; i++) {
			h[i] = c_IV256[i];
		}

#if !USE_CACHE
		blake256_compress(h, c_data, 512);
#else
		if (crcsum != prevsum) {
			prevsum = crcsum;
			blake256_compress(h, c_data, 512, rounds);
			#pragma unroll
			for(int i=0; i<8; i++) {
				cache[i] = h[i];
			}
		} else {
			#pragma unroll
			for(int i=0; i<8; i++) {
				h[i] = cache[i];
			}
		}
#endif
		// ------ Close: Bytes 64 to 80 ------ 

		uint32_t ending[4];
		ending[0] = c_data[16];
		ending[1] = c_data[17];
		ending[2] = c_data[18];
		ending[3] = nounce; /* our tested value */

		blake256_compress(h, ending, 640, rounds);

		// not sure why, h[7] is ok
		h[6] = cuda_swab32(h[6]);

		// compare count of leading zeros h[6] + h[7]
		uint64_t high64 = ((uint64_t*)h)[3];
		if (high64 <= highTarget)
#if NBN == 2
		/* keep the smallest nounce, + extra one if found */
		if (resNounce[0] > nounce) {
			// printf("%llx %llx \n", high64, highTarget);
			resNounce[1] = resNounce[0];
			resNounce[0] = nounce;
		}
		else
			resNounce[1] = nounce;
#else
		resNounce[0] = nounce;
#endif
	}
}

__host__
uint32_t blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint64_t highTarget,
	const uint32_t crcsum, const int8_t rounds)
{
	const int threadsperblock = TPB;
	uint32_t result = MAXU;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	size_t shared_size = 0;

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t)) != hipSuccess)
		return result;

	blake256_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNonce, d_resNonce[thr_id], highTarget, crcsum, (int) rounds);
	hipDeviceSynchronize();
	if (hipSuccess == hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		//hipDeviceSynchronize(); /* seems no more required */
		result = h_resNonce[thr_id][0];
		for (int n=0; n < (NBN-1); n++)
			extra_results[n] = h_resNonce[thr_id][n+1];
	}
	return result;
}

__host__
void blake256_cpu_setBlock_80(uint32_t *pdata, const uint32_t *ptarget)
{
	uint32_t data[20];
	memcpy(data, pdata, 80);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, sizeof(data), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_sigma), host_sigma, sizeof(host_sigma), 0, hipMemcpyHostToDevice));
}

extern "C" int scanhash_blake256(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done, int8_t blakerounds=14)
{
	const uint32_t first_nonce = pdata[19];
	static bool init[8] = { 0, 0, 0, 0, 0, 0, 0, 0 };
	uint32_t throughput = min(TPB * 4096, max_nonce - first_nonce);
	uint64_t targetHigh = ((uint64_t*)ptarget)[3];
	uint32_t crcsum = MAXU;
	int rc = 0;

#if NBN > 1
	if (extra_results[0] != MAXU) {
		// possible extra result found in previous call
		if (first_nonce <= extra_results[0] && max_nonce >= extra_results[0]) {
			pdata[19] = extra_results[0];
			*hashes_done = pdata[19] - first_nonce + 1;
			extra_results[0] = MAXU;
			rc = 1;
			goto exit_scan;
		}
	}
#endif

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00000f;

	if (!init[thr_id]) {
		if (opt_n_threads > 1) {
			CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		}
		CUDA_SAFE_CALL(hipHostMalloc(&h_resNonce[thr_id], NBN * sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)));
		init[thr_id] = true;
	}

	blake256_cpu_setBlock_80(pdata, ptarget);
#if USE_CACHE
	crcsum = crc32_u32t(pdata, 64);
#endif

	do {
		// GPU HASH
		uint32_t foundNonce = blake256_cpu_hash_80(thr_id, throughput, pdata[19], targetHigh, crcsum, blakerounds);
		if (foundNonce != MAXU)
		{
			uint32_t endiandata[20];
			uint32_t vhashcpu[8];
			uint32_t Htarg = ptarget[6];

			for (int k=0; k < 19; k++)
				be32enc(&endiandata[k], pdata[k]);

			be32enc(&endiandata[19], foundNonce);

			blake256hash(vhashcpu, endiandata, blakerounds);

			if (vhashcpu[6] <= Htarg || cuda_swab32(vhashcpu[6]) <= Htarg /*&& fulltest(vhashcpu, ptarget)*/)
			{
				pdata[19] = foundNonce;
				rc = 1;

				if (extra_results[0] != MAXU) {
					// Rare but possible if the throughput is big
					be32enc(&endiandata[19], extra_results[0]);
					blake256hash(vhashcpu, endiandata, blakerounds);
					if (vhashcpu[6] <= Htarg /* && fulltest(vhashcpu, ptarget) */) {
						applog(LOG_NOTICE, "GPU found more than one result " CL_GRN "yippee!");
						rc = 2;
					} else {
						extra_results[0] = MAXU;
					}
				}

				//applog_hash((uint8_t*)ptarget);
				//applog_compare_hash((uint8_t*)vhashcpu,(uint8_t*)ptarget);
				goto exit_scan;
			}
			else if (opt_debug) {
				applog_hash((uint8_t*)ptarget);
				applog_compare_hash((uint8_t*)vhashcpu,(uint8_t*)ptarget);
				applog(LOG_DEBUG, "GPU #%d: result for nounce %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}

		if ((uint64_t) pdata[19] + throughput > (uint64_t) max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

exit_scan:
	*hashes_done = pdata[19] - first_nonce + 1;
	return rc;
}
